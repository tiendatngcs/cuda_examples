
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N (1 << 16)
#define N_FLOAT (N*sizeof(float))


// Code written in C 
// void vector_add(float *out, float *a, float *b, int n) {
//     for(int i = 0; i < n; i++){
//         out[i] = a[i] + b[i];
//     }
// }

// int main(){
//     float *a, *b, *out; 

//     // Allocate memory
//     a   = (float*)malloc(sizeof(float) * N);
//     b   = (float*)malloc(sizeof(float) * N);
//     out = (float*)malloc(sizeof(float) * N);

//     // Initialize array
//     for(int i = 0; i < N; i++){
//         a[i] = 1.0f; b[i] = 2.0f;
//     }

//     // Main function
//     vector_add(out, a, b, N);
// }

// ======== We now convert to CUDA code ========

__global__ void vector_add_kernel(float* out, float* a, float* b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    // allocate memory on CPU
    float* a = (float*)malloc(sizeof(float) * N);
    float* b = (float*)malloc(sizeof(float) * N);
    float* out = (float*)malloc(sizeof(float) * N);

    // init arrays
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
        out[i] = 0;
    }

    // allocate GPU memory
    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, N_FLOAT);
    hipMalloc(&d_b, N_FLOAT);
    hipMalloc(&d_out, N_FLOAT);

    // Copy the numbers over to device
    hipMemcpy(d_a, a, N_FLOAT, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N_FLOAT, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, N_FLOAT, hipMemcpyHostToDevice);

    // do vector addition
    vector_add_kernel<<<1, 1>>>(d_out, d_a, d_b, N);

    // copy back to host 
    hipMemcpy(out, d_out, N_FLOAT, hipMemcpyDeviceToHost);

    printf("Value of C %lf\n", out[0]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);


}